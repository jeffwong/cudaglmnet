#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/generate.h>

#define DEBUG 0

typedef struct {
    int n,p,num_lambda;
    float* lambda;
    thrust::device_ptr<float> X, y;
} data;

typedef struct {
    thrust::device_ptr<float> beta, beta_old, theta, theta_old, momentum;
} coef;

typedef struct {
    float nLL;
    thrust::device_ptr<float> eta, yhat, residuals, grad, U, diff_beta, diff;
} opt;

typedef struct {
    int type, maxIt, reset;
    float gamma, t, thresh;
} misc;

struct square
{
    __host__ __device__
        float operator()(const float& x) const { 
            return x*x;
        }
};

struct soft_threshold
{
    const float lambda;

    soft_threshold(float _lambda) : lambda(_lambda) {}

    __host__ __device__
        float operator()(const float& x) const { 
            if (x > -lambda && x < lambda) return 0;
            else if (x > lambda) return x - lambda;
            else return x + lambda;
        }
};

struct saxpy
{
    const float a;

    saxpy(float _a) : a(_a) {}

    __host__ __device__
        float operator()(const float& x, const float& y) const { 
            return a * x + y;
        }
};

struct absolute_value
{
    __host__ __device__
        float operator()(const float& x) const { 
            if (x < 0) return (-1*x);
            else return x;
        }
};

void printDevPtr(thrust::device_ptr<float> x, int size)
{
  thrust::device_vector<float> d(x, x+size);
  int i = 0;
  for (i = 0; i < size; i++) std::cout << "D[" << i << "] = " << d[i] << std::endl; 
}

void init(data*, coef*, opt*, misc*,
          float*, float*, int, int, float*, int,
          int, float*, int, float, float,
          float, int);
void pathSol(data*, coef*, opt*, misc*, float*, hipblasStatus_t, hipblasHandle_t);
void singleSolve(data*, coef*, opt*, misc*, int, hipblasStatus_t, hipblasHandle_t);
float calcNegLL(data*, coef*, opt*, misc*, thrust::device_ptr<float>, int, hipblasStatus_t, hipblasHandle_t);
void gradStep(data*, coef*, opt*, misc*, int, hipblasStatus_t, hipblasHandle_t);
void proxCalc(data*, coef*, opt*, misc*, int, hipblasStatus_t, hipblasHandle_t);
void nestStep(data*, coef*, opt*, misc*, int, int, hipblasStatus_t, hipblasHandle_t);
int checkStep(data*, coef*, opt*, misc*, int, hipblasStatus_t, hipblasHandle_t handle );
int checkCrit(data*, coef*, opt*, misc*, int, int, hipblasStatus_t, hipblasHandle_t);
void shutdown(data*, coef*, opt*, misc*);
void device_ptr2Norm(thrust::device_ptr<float>, float*, int, hipblasStatus_t, hipblasHandle_t);
void device_ptrDot(thrust::device_ptr<float>, thrust::device_ptr<float>,
                   float*, int, hipblasStatus_t, hipblasHandle_t);
float device_ptrMaxNorm(thrust::device_ptr<float>, int);
void device_ptrSoftThreshold(thrust::device_ptr<float>, thrust::device_ptr<float>, float, int);
void device_ptrSgemv(thrust::device_ptr<float>,
                          thrust::device_ptr<float>,
                          thrust::device_ptr<float>,
                          int, int,
                          hipblasStatus_t, hipblasHandle_t);
void device_ptrCrossProd(thrust::device_ptr<float>,
                         thrust::device_ptr<float>,
                         thrust::device_ptr<float>,
                         int, int,
                         hipblasStatus_t, hipblasHandle_t) ;
thrust::device_ptr<float> makeDeviceVector(float*, int);
thrust::device_ptr<float> makeEmptyDeviceVector(int);
void device_ptrCopy(thrust::device_ptr<float>,
                    thrust::device_ptr<float>,
                    int);

  void init(data* ddata, coef* dcoef, opt* dopt, misc* dmisc,
            float* X, float* y, int n, int p, float* lambda, int num_lambda,
            int type, float* beta, int maxIt, float thresh, float gamma,
            float t, int reset)
  {
    if (DEBUG) printf("Inside init\n");

    /* Set data variables */
    ddata->lambda = lambda;
    ddata->n = n;
    ddata->p = p;
    ddata->num_lambda = num_lambda;

    dopt->nLL = 0;

    /* Set misc variables */

    dmisc->type = type;
    dmisc->maxIt = maxIt;
    dmisc->gamma = gamma;
    dmisc->t = t;
    dmisc->reset = reset;
    dmisc->thresh = thresh;
  }

  void pathSol(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, float* beta,
               hipblasStatus_t stat, hipblasHandle_t handle)
  {
    if (DEBUG) printf("Inside pathSol\n");
    int j = 0;
    for (j=0; j < ddata->num_lambda; j++){
      //beta_old is never used
      //device_ptrCopy(dcoef->beta, dcoef->beta_old, ddata->p);
      device_ptrCopy(dcoef->theta, dcoef->theta_old, ddata->p);
      singleSolve(ddata, dcoef, dopt, dmisc, j, stat, handle);
    }
    hipMemcpy(beta, thrust::raw_pointer_cast(dcoef->beta),
               sizeof(float) * (ddata->num_lambda * ddata->p), hipMemcpyDeviceToHost);
  }

  void singleSolve(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, int j,
                   hipblasStatus_t stat, hipblasHandle_t handle)
  {
    if (DEBUG) printf("Inside singleSolve\n");
    int iter = 0;
    do
    {
      calcNegLL(ddata, dcoef, dopt, dmisc, dcoef->beta, j, stat, handle);
      do
      {
        gradStep(ddata, dcoef, dopt, dmisc, j, stat, handle);
      } while (checkStep(ddata, dcoef, dopt, dmisc, j, stat, handle) == 0);
      nestStep(ddata, dcoef, dopt, dmisc, j, iter, stat, handle);
      iter = iter + 1;
    } while (checkCrit(ddata, dcoef, dopt, dmisc, j, iter, stat, handle) == 0);
  }

  float calcNegLL(data* ddata, coef* dcoef, opt* dopt, misc* dmisc,
                  thrust::device_ptr<float> pvector, int j,
                  hipblasStatus_t stat, hipblasHandle_t handle)
  {
    if (DEBUG) printf("Inside calcNegLL\n");

    device_ptrSgemv(ddata->X, pvector, dopt->eta, ddata->n, ddata->p, stat, handle);
    switch (dmisc->type)
    {
      case 0:  //normal
      {
        float nll = 0;
        device_ptr2Norm(dopt->residuals, &nll, ddata->n, stat, handle);
        dopt->nLL = 0.5 * nll;
        break;
      }
      default:  //default to normal
      { 
        float nll = 0;
        device_ptr2Norm(dopt->residuals, &nll, ddata->n, stat, handle);
        dopt->nLL = 0.5 * nll;
        break;
      }
    }
    if (DEBUG) printf("calcNegLL nll %f\n", dopt->nLL);
    return dopt->nLL;
  }

  void gradStep(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, int j,
                hipblasStatus_t stat, hipblasHandle_t handle)
  {
    if (DEBUG) printf("Inside gradStep\n");
    switch (dmisc->type)
    {
      case 0:  //normal
      {
        //yhat = XB
        device_ptrSgemv(ddata->X, dcoef->beta, dopt->yhat, ddata->n, ddata->p, stat, handle);
        //residuals = y - yhat
        thrust::transform(ddata->y, ddata->y + ddata->n,
                          dopt->yhat,
                          dopt->residuals,
                          thrust::minus<float>());
        hipDeviceSynchronize();
        //grad = -X^T residuals
        device_ptrCrossProd(ddata->X, dopt->residuals, dopt->grad, ddata->n,
                            ddata->p, stat, handle);
        thrust::device_vector<float> ones(ddata->p, -1);
        thrust::transform(dopt->grad, dopt->grad + ddata->p,
                          ones.begin(), dopt->grad,
                          thrust::multiplies<float>());
        //U = -t * grad + beta
        thrust::transform(dopt->grad, dopt->grad + ddata->p,
                          dcoef->beta,
                          dopt->U,
                          saxpy(-dmisc->t));
        hipDeviceSynchronize();
        proxCalc(ddata, dcoef, dopt, dmisc, j, stat, handle);
        hipDeviceSynchronize();
        break;
      }
      default:
      {
        break;
      }
    } 
  }

  void proxCalc(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, int j,
                hipblasStatus_t stat, hipblasHandle_t handle)
  {
    if (DEBUG) printf("Inside proxCalc\n");
    switch (dmisc->type)
    {
      case 0:  //normal
      {
        device_ptrSoftThreshold(dopt->U, dcoef->theta, ddata->lambda[j] * dmisc->t, ddata->p);
        break;
      }
      default:
      {
        device_ptrSoftThreshold(dopt->U, dcoef->theta, ddata->lambda[j] * dmisc->t, ddata->p);
        break;
      }
    }
  }

  int checkStep(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, int j,
                hipblasStatus_t stat, hipblasHandle_t handle)
  {
    if (DEBUG) printf("Inside checkStep\n");
    float nLL = calcNegLL(ddata, dcoef, dopt, dmisc, dcoef->theta, j, stat, handle);
    
    //diff = theta - beta
    thrust::transform(dcoef->theta, dcoef->theta + ddata->p,
                      dcoef->beta,
                      dopt->diff,
                      thrust::minus<float>());
    //iprod is the dot product of diff and grad
    float iprod=0; device_ptrDot(dopt->diff, dopt->grad, &iprod, ddata->p, stat, handle);
    float sumSquareDiff=0; device_ptr2Norm(dopt->diff, &sumSquareDiff, ddata->p, stat, handle);

    int check = (int)(nLL < ((dopt->nLL + iprod + sumSquareDiff) / (2 * dmisc->t)));
    if (check == 0) dmisc->t = dmisc->t * dmisc->gamma;
    return check;
  }

  void nestStep(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, int j, int iter,
                hipblasStatus_t stat, hipblasHandle_t handle)
  {
    if (DEBUG) printf("Inside nestStep\n");
    //beta_old is never used
    //device_ptrCopy(dcoef->beta, dcoef->beta_old, ddata->p);
    //momentum = theta - theta old
    thrust::transform(dcoef->theta, dcoef->theta + ddata->p,
                      dcoef->theta_old,
                      dcoef->momentum,
                      thrust::minus<float>());
    float scale = ((float) (iter % dmisc->reset)) / (iter % dmisc->reset + 3);
    //beta = theta + scale*momentum
    thrust::transform(dcoef->momentum, dcoef->momentum + ddata->p,
                      dcoef->theta,
                      dcoef->beta,
                      saxpy(scale));
    device_ptrCopy(dcoef->theta, dcoef->theta_old, ddata->p);
  }

  int checkCrit(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, int j, int iter,
                hipblasStatus_t stat, hipblasHandle_t handle)
  {
    if (DEBUG) printf("Inside checkCrit\n");
    float move = device_ptrMaxNorm(dopt->diff, ddata->p); 
    if (DEBUG) printf("move %f\n", move);
    return ((iter > dmisc->maxIt) || (move < dmisc->thresh));
  }

  void shutdown(data* ddata, coef* dcoef, opt* dopt, misc* dmisc)
  {
    free(ddata); free(dcoef); free(dopt); free(dmisc);
  }

  /*
    MISC MATH FUNCTIONS
  */

  void device_ptrCopy(thrust::device_ptr<float> from,
                      thrust::device_ptr<float> to,
                      int size)
  {
    hipMemcpy(thrust::raw_pointer_cast(&to[0]), thrust::raw_pointer_cast(&from[0]),
               sizeof(float) * size, hipMemcpyDeviceToDevice);
  }

  // ||x||_max
  float device_ptrMaxNorm(thrust::device_ptr<float> x, int size)
  {
    return thrust::transform_reduce(x, x + size,
                                    absolute_value(), (float) 0, thrust::maximum<float>());  
  }

  // ||x||_2^2
  void device_ptr2Norm(thrust::device_ptr<float> x, float* result, int size,
                       hipblasStatus_t stat, hipblasHandle_t handle)
  {  
    hipblasSnrm2(handle, size, thrust::raw_pointer_cast(x), 1, result);
    hipDeviceSynchronize();
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      printf ("CUBLAS snrm2 failed with error %i\n", stat);
    }
  }

  void device_ptrDot(thrust::device_ptr<float> x, thrust::device_ptr<float> y,
                     float* result, int size,
                     hipblasStatus_t stat, hipblasHandle_t handle)
  {  
    hipblasSdot(handle, size, thrust::raw_pointer_cast(x), 1,
               thrust::raw_pointer_cast(y), 1, result);
    hipDeviceSynchronize();
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      printf ("CUBLAS sdot failed with error %i\n", stat);
    }
  }

  // b = X^T y
  void device_ptrCrossProd(thrust::device_ptr<float> X,
                           thrust::device_ptr<float> y,
                           thrust::device_ptr<float> b,
                           int n, int p,
                           hipblasStatus_t stat, hipblasHandle_t handle)
  {
    float alpha = 1; float beta = 0;
    stat = hipblasSgemv(handle, HIPBLAS_OP_T, n, p, &alpha,
                thrust::raw_pointer_cast(&X[0]), n,
                thrust::raw_pointer_cast(&y[0]), 1,
                &beta, thrust::raw_pointer_cast(&b[0]), 1);
    hipDeviceSynchronize();
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      printf ("CrossProd using CUBLAS sgemv failed with error %i\n", stat);
    }
  }

  // b = Ax
  void device_ptrSgemv(thrust::device_ptr<float> A,
                       thrust::device_ptr<float> x,
                       thrust::device_ptr<float> b,
                       int n, int p,
                       hipblasStatus_t stat, hipblasHandle_t handle)
  {
    float alpha = 1; float beta = 0;
    stat = hipblasSgemv(handle, HIPBLAS_OP_N, n, p, &alpha,
                       thrust::raw_pointer_cast(&A[0]), n,
                       thrust::raw_pointer_cast(&x[0]), 1,
                       &beta, thrust::raw_pointer_cast(&b[0]), 1);
    hipDeviceSynchronize();
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      printf ("CUBLAS sgemv failed with error %i\n", stat);
    }
  }

  // S(x, lambda)
  void device_ptrSoftThreshold(thrust::device_ptr<float> x,
                               thrust::device_ptr<float> dest,
                               float lambda, int size)
  {
    thrust::transform(x, x + size,
                      dest, soft_threshold(lambda));
    hipDeviceSynchronize();
  }
  
extern "C"{

  /*
    Entry point for R
    X is a matrix (represented as a 1d array) that is n by p
    y is a vector that is n by 1
  */
  void activePathSol(float* X, float* y, int* n, int* p, float* lambda, int* num_lambda,
                     int* type, float* beta, int* maxIt, float* thresh, float* gamma,
                     float* t, int* reset)
  { 
    //setup pointers
    data* ddata = (data*)malloc(sizeof(data));
    coef* dcoef = (coef*)malloc(sizeof(coef));
    opt* dopt = (opt*)malloc(sizeof(opt));
    misc* dmisc = (misc*)malloc(sizeof(misc));

    /* Set key data variables X, y, beta */

    thrust::device_vector<float> dX(X, X+(n[0]*p[0]));
    thrust::device_vector<float> dy(y, y+n[0]);
    thrust::device_vector<float> dbeta(beta, beta+p[0]);
    //beta_old is never used
    //thrust::device_vector<float> dbeta_old(beta, beta+p[0]);

    ddata->X = dX.data();
    ddata->y = dy.data();
    dcoef->beta = dbeta.data();
    //beta_old is never used
    //dcoef->beta_old = dbeta_old.data();

    /* Set coef variables */

    thrust::device_vector<float> dtheta(p[0],0);
    thrust::device_vector<float> dtheta_old(p[0],0);
    thrust::device_vector<float> dmomentum(p[0],0);
    dcoef->theta = dtheta.data();
    dcoef->theta_old = dtheta_old.data();
    dcoef->momentum = dmomentum.data();

    /* Set optimization variables */

    thrust::device_vector<float> deta(n[0],0);
    thrust::device_vector<float> dyhat(n[0],0);
    thrust::device_vector<float> dresiduals(n[0],0);
    thrust::device_vector<float> dgrad(p[0],0);
    thrust::device_vector<float> dU(p[0],0);
    //beta_old and diff_beta are never used
    //thrust::device_vector<float> ddiff_beta(p[0],0);
    thrust::device_vector<float> ddiff(p[0],0);
    dopt->eta = deta.data();
    dopt->yhat = dyhat.data();
    dopt->residuals = dresiduals.data();
    dopt->grad = dgrad.data();
    dopt->U = dU.data();
    //beta_old and diff_beta are never used
    //dopt->diff_beta = ddiff_beta.data();
    dopt->diff = ddiff.data();

    //allocate pointers
    init(ddata, dcoef, dopt, dmisc,
         X, y, *n, *p, lambda, *num_lambda,
         *type, beta, *maxIt, *thresh, *gamma,
         *t, *reset);
   
    //Set cublas variables
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);

    //solve
    device_ptrSgemv(ddata->X, dcoef->beta, dopt->yhat, ddata->n, ddata->p, stat, handle);
    thrust::transform(ddata->y, ddata->y + ddata->n,
                          dopt->yhat,
                          dopt->residuals,
                          thrust::minus<float>());
    pathSol(ddata, dcoef, dopt, dmisc, beta, stat, handle);
    //shutdown*/
    shutdown(ddata, dcoef, dopt, dmisc);
    hipblasDestroy(handle);
  }

}

int main() {
  int* n = (int*)malloc(sizeof(int)); n[0] = 100;
  int* p = (int*)malloc(sizeof(int)); p[0] = 10;
  int* num_lambda = (int*)malloc(sizeof(int)); num_lambda[0] = 1;
 
  thrust::host_vector<float> X(n[0]*p[0],1);
  thrust::host_vector<float> y(n[0],1);
  thrust::host_vector<float> beta(p[0] * num_lambda[0],1);
  thrust::sequence(X.begin(), X.end());
  thrust::sequence(y.begin(), y.end());
  
  int* type = (int*)malloc(sizeof(int)); type[0] = 0;
  int* maxIt = (int*)malloc(sizeof(int)); maxIt[0] = 10;
  int* reset = (int*)malloc(sizeof(int)); reset[0] = 30;
  float* lambda = (float*)malloc(sizeof(float) * num_lambda[0]); lambda[0] = 1;
  float* thresh = (float*)malloc(sizeof(float)); thresh[0] = 0.00001;
  float* gamma = (float*)malloc(sizeof(float)); gamma[0] = 0.9;
  float* t = (float*)malloc(sizeof(float)); t[0] = 10;

  activePathSol(thrust::raw_pointer_cast(&X[0]),
                thrust::raw_pointer_cast(&y[0]),
                n, p, lambda, num_lambda,
                type, thrust::raw_pointer_cast(&beta[0]), maxIt, thresh, gamma,
                t, reset);
  int i = 0;
  for(i = 0; i < beta.size(); i++) printf("beta[%i]: %f\n", i, beta[i]); 
  free(n); free(p); free(num_lambda);
  free(type); free(maxIt); free(reset);
  free(lambda); free(thresh); free(gamma); free(t);
  return 0;
}
