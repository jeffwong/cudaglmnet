#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/copy.h>
#include <thrust/fill.h>

typedef struct {
    int n,p,num_lambda;
    thrust::host_vector<float> lambda;
    thrust::device_vector<float> X, y;
} data;

typedef struct {
    thrust::device_vector<float> beta, beta_old, theta, theta_old, momentum;
} coef;

typedef struct {
    float nLL;
    thrust::device_vector<float> eta, yhat, residuals, grad, U, diff_beta, diff_theta;
} opt;

typedef struct {
    int type, maxIt, reset;
    float gamma, t, thresh;
} misc;

struct square
{
    __host__ __device__
        float operator()(const float& x) const { 
            return x*x;
        }
};

struct soft_threshold
{
    const float lambda;

    soft_threshold(float _lambda) : lambda(_lambda) {}

    __host__ __device__
        float operator()(const float& x) const { 
            if (x > -lambda && x < lambda) return 0;
            else if (x > lambda) return x - lambda;
            else return x + lambda;
        }
};

struct saxpy
{
    const float a;

    saxpy(float _a) : a(_a) {}

    __host__ __device__
        float operator()(const float& x, const float& y) const { 
            return a * x + y;
        }
};

struct absolute_value
{
    __host__ __device__
        float operator()(const float& x) const { 
            if (x < 0) return (-1*x);
            else return x;
        }
};


  
extern "C"{


void activePathSol(float*, float*, int*, int*, float*, int*,
                   int*, float*, int*, float*, float*,
                   float*, int*);
void init(data*, coef*, opt*, misc*,
          float*, float*, int, int, float*, int,
          int, float*, int, float, float,
          float, int);
void pathSol(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, float* beta,hipblasStatus_t stat, hipblasHandle_t handle );
void singleSolve(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, int j,hipblasStatus_t stat, hipblasHandle_t handle );
float calcNegLL(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, thrust::device_vector<float> pvector, int j,hipblasStatus_t stat, hipblasHandle_t handle );
void gradStep(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, int j,hipblasStatus_t stat, hipblasHandle_t handle );
void proxCalc(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, int j,hipblasStatus_t stat, hipblasHandle_t handle );
void nestStep(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, int j, int iter,hipblasStatus_t stat, hipblasHandle_t handle );
int checkStep(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, int j,hipblasStatus_t stat, hipblasHandle_t handle );
int checkCrit(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, int j, int iter,hipblasStatus_t stat, hipblasHandle_t handle );
void shutdown(data* ddata, coef* dcoef, opt* dopt, misc* dmisc);
void device_vector2Norm(thrust::device_vector<float> x, float* result, hipblasStatus_t stat, hipblasHandle_t handle );
void device_vectorDot(thrust::device_vector<float> x, thrust::device_vector<float> y,
                      float* result, hipblasStatus_t stat, hipblasHandle_t handle );
float device_vectorMaxNorm(thrust::device_vector<float> x);
void device_vectorSoftThreshold(thrust::device_vector<float> x, thrust::device_vector<float>, float lambda);
void device_vectorSgemv(thrust::device_vector<float> A,
                          thrust::device_vector<float> x,
                          thrust::device_vector<float> b,
                          int n, int p,
                          hipblasStatus_t stat, hipblasHandle_t handle );
void device_vectorCrossProd(thrust::device_vector<float> X,
                              thrust::device_vector<float> y,
                              thrust::device_vector<float> b,
                              int n, int p,
                            hipblasStatus_t stat, hipblasHandle_t handle) ;
thrust::device_vector<float> makeDeviceVector(float* x, int size);
thrust::device_vector<float> makeEmptyDeviceVector(int size);
 


  /*
    Entry point for R
    X is a matrix (represented as a 1d array) that is n by p
    y is a vector that is n by 1
  */
  void activePathSol(float* X, float* y, int* n, int* p, float* lambda, int* num_lambda,
                     int* type, float* beta, int* maxIt, float* thresh, float* gamma,
                     float* t, int* reset)
  { 
    //setup pointers
    data* ddata = (data*)malloc(sizeof(data));
    coef* dcoef = (coef*)malloc(sizeof(coef));
    opt* dopt = (opt*)malloc(sizeof(opt));
    misc* dmisc = (misc*)malloc(sizeof(misc));
 
    //allocate pointers, init cublas
    init(ddata, dcoef, dopt, dmisc,
         X, y, n[0], p[0], lambda, num_lambda[0],
         type[0], beta, maxIt[0], thresh[0], gamma[0],
         t[0], reset[0]);
    
    //Set cublas variables
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      printf ("CUBLAS initialization failed\n");
      return;
    }
 
    //solve
    pathSol(ddata, dcoef, dopt, dmisc, beta, stat, handle);

    //shutdown
    shutdown(ddata, dcoef, dopt, dmisc);
    hipblasDestroy(handle);
  }

  void init(data* ddata, coef* dcoef, opt* dopt, misc* dmisc,
            float* X, float* y, int n, int p, float* lambda, int num_lambda,
            int type, float* beta, int maxIt, float thresh, float gamma,
            float t, int reset)
  {
    /* Set data variables */

    ddata->X = makeDeviceVector(X, n*p);
    ddata->y = makeDeviceVector(y, n);
    ddata->lambda = thrust::host_vector<float>(lambda, lambda+num_lambda);
    ddata->n = n;
    ddata->p = p;
    ddata->num_lambda = num_lambda;

    /* Set coef variables */

    dcoef->beta = makeDeviceVector(beta, p);
    dcoef->beta_old = makeDeviceVector(beta, p);
    dcoef->theta = makeEmptyDeviceVector(p);
    dcoef->theta_old = makeEmptyDeviceVector(p);
    dcoef->momentum = makeEmptyDeviceVector(p);

    /* Set optimization variables */

    dopt->nLL = 0;
    dopt->eta = makeEmptyDeviceVector(n);
    dopt->yhat = makeEmptyDeviceVector(n);
    dopt->residuals = makeEmptyDeviceVector(n);
    dopt->grad = makeEmptyDeviceVector(p);
    dopt->U = makeEmptyDeviceVector(p);
    dopt->diff_beta = makeEmptyDeviceVector(p);
    dopt->diff_theta = makeEmptyDeviceVector(p);

    /* Set misc variables */

    dmisc->type = type;
    dmisc->maxIt = maxIt;
    dmisc->gamma = gamma;
    dmisc->t = t;
    dmisc->reset = reset;
    dmisc->thresh = thresh;
  }

  void pathSol(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, float* beta,
               hipblasStatus_t stat, hipblasHandle_t handle)
  {
    int j = 0;
    for (j=0; j < ddata->num_lambda; j++){
      dcoef->beta_old = dcoef->beta;
      dcoef->theta_old = dcoef->theta;
      singleSolve(ddata, dcoef, dopt, dmisc, j, stat, handle);

      int startIndex = j*ddata->p;
      thrust::copy(dcoef->beta.begin(), dcoef->beta.end(), beta + startIndex);
    }
  }

  void singleSolve(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, int j,
                   hipblasStatus_t stat, hipblasHandle_t handle)
  {
    int iter = 0;
    do
    {
      printf("calcNegLL\n");
      calcNegLL(ddata, dcoef, dopt, dmisc, dcoef->beta, j, stat, handle);
      do
      {
        printf("gradStep\n");
        gradStep(ddata, dcoef, dopt, dmisc, j, stat, handle);
        printf("checkStep\n");
      } while (checkStep(ddata, dcoef, dopt, dmisc, j, stat, handle) == 0);
      printf("nestStep\n");
      nestStep(ddata, dcoef, dopt, dmisc, j, iter, stat, handle);
      iter = iter + 1;
      printf("checkCrit");
    } while (checkCrit(ddata, dcoef, dopt, dmisc, j, iter, stat, handle));
  }

  int checkCrit(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, int j, int iter,
                hipblasStatus_t stat, hipblasHandle_t handle)
  {
    float move = device_vectorMaxNorm(dopt->diff_theta);  
    if ((iter > dmisc->maxIt) || (move < dmisc->thresh)) return 0;
    else return 1;
  }

  float calcNegLL(data* ddata, coef* dcoef, opt* dopt, misc* dmisc,
                  thrust::device_vector<float> pvector, int j,
                  hipblasStatus_t stat, hipblasHandle_t handle)
  {
    device_vectorSgemv(ddata->X, pvector, dopt->eta, ddata->n, ddata->p, stat, handle);
    switch (dmisc->type)
    {
      case 0:  //normal
      {
        float nll = 0;
        device_vector2Norm(dopt->residuals, &nll, stat, handle);
        dopt->nLL = 0.5 * nll;
        break;
      }
      default:  //default to normal
      { 
        float nll = 0;
        device_vector2Norm(dopt->residuals, &nll, stat, handle);
        dopt->nLL = 0.5 * nll;
        break;
      }
    }
    return dopt->nLL;
  }

  int checkStep(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, int j,
                hipblasStatus_t stat, hipblasHandle_t handle)
  {
    float nLL = calcNegLL(ddata, dcoef, dopt, dmisc, dcoef->theta, j, stat, handle);
    
    //iprod is the dot product of diff and grad
    float iprod=0; device_vectorDot(dopt->diff_theta, dopt->grad, &iprod, stat, handle);
    float sumSquareDiff=0; device_vector2Norm(dopt->diff_theta, &sumSquareDiff, stat, handle);

    int check = (int)(nLL < (dopt->nLL + iprod + sumSquareDiff) / (2 * dmisc->t));
    if (check == 0) dmisc->t = dmisc->t * dmisc->gamma;
    return check;
  }

  void gradStep(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, int j,
                hipblasStatus_t stat, hipblasHandle_t handle)
  {
    switch (dmisc->type)
    {
      case 0:  //normal
      {
        //yhat = XB
        device_vectorSgemv(ddata->X, dcoef->beta, dopt->yhat, ddata->n, ddata->p, stat, handle);
        //residuals = y - yhat
        thrust::transform(ddata->y.begin(), ddata->y.end(),
                          dopt->yhat.begin(),
                          dopt->residuals.begin(),
                          thrust::minus<float>());
        //grad = X^T residuals
        device_vectorCrossProd(ddata->X, dopt->residuals, dopt->grad, ddata->n,
                               ddata->p, stat, handle);
        //U = -t * grad + beta
        thrust::transform(dopt->grad.begin(), dopt->grad.end(),
                          dcoef->beta.begin(),
                          dopt->U.begin(),
                          saxpy(-dmisc->t));
        proxCalc(ddata, dcoef, dopt, dmisc, j, stat, handle);
        break;
      }
      default:
      {
        break;
      }
    } 
  }

  void proxCalc(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, int j,
                hipblasStatus_t stat, hipblasHandle_t handle)
  {
    switch (dmisc->type)
    {
      case 0:  //normal
      {
        device_vectorSoftThreshold(dopt->U, dcoef->theta, ddata->lambda[j] * dmisc->t);
        break;
      }
      default:
      {
        break;
      }
    }
  }

  void nestStep(data* ddata, coef* dcoef, opt* dopt, misc* dmisc, int j, int iter,
                hipblasStatus_t stat, hipblasHandle_t handle)
  {
    dcoef->beta_old = dcoef->beta;
    //momentum = theta - theta old
    thrust::transform(dcoef->theta.begin(), dcoef->theta.end(),
                      dcoef->theta_old.begin(),
                      dcoef->momentum.begin(),
                      thrust::minus<float>());
    float scale = (float) (iter % dmisc->reset) / (iter % dmisc->reset + 3);
    //beta = theta + scale*momentum
    thrust::transform(dcoef->momentum.begin(), dcoef->momentum.end(),
                      dcoef->theta.begin(),
                      dcoef->beta.begin(),
                      saxpy(scale));
    dcoef->theta_old = dcoef->theta;
  }

  void shutdown(data* ddata, coef* dcoef, opt* dopt, misc* dmisc)
  {
    free(ddata); free(dcoef); free(dopt); free(dmisc);
  }

  /*
    MISC MATH FUNCTIONS
  */

  thrust::device_vector<float> makeDeviceVector(float* x, int size)
  {
    return thrust::device_vector<float> (x, x+size);
  }

  thrust::device_vector<float> makeEmptyDeviceVector(int size)
  {
    thrust::host_vector<float> x(size, 0);
    thrust::device_vector<float> dx = x;
    return dx;
  }

  // ||x||_max
  float device_vectorMaxNorm(thrust::device_vector<float> x)
  {
    return thrust::transform_reduce(x.begin(), x.end(),
                                    absolute_value(), 0.0, thrust::maximum<float>());  
  }

  // ||x||_2^2
  void device_vector2Norm(thrust::device_vector<float> x, float* result, hipblasStatus_t stat, hipblasHandle_t handle)
  {  
    hipblasSnrm2(handle, x.size(), thrust::raw_pointer_cast(&x[0]), 1, result);
  }

  void device_vectorDot(thrust::device_vector<float> x, thrust::device_vector<float> y,
                         float* result,
                         hipblasStatus_t stat, hipblasHandle_t handle)
  {  
    hipblasSdot(handle, x.size(), thrust::raw_pointer_cast(&x[0]), 1,
               thrust::raw_pointer_cast(&y[0]), 1, result);
  }

  // b = X^T y
  void device_vectorCrossProd(thrust::device_vector<float> X,
                              thrust::device_vector<float> y,
                              thrust::device_vector<float> b,
                              int n, int p,
                              hipblasStatus_t stat, hipblasHandle_t handle)
  {
    float alpha = 1; float beta = 0;
    hipblasSgemv(handle, HIPBLAS_OP_T, n, p, &alpha,
                thrust::raw_pointer_cast(&X[0]), n,
                thrust::raw_pointer_cast(&y[0]), 1,
                &beta, thrust::raw_pointer_cast(&b[0]), 1); 
  }

  // b = Ax
  void device_vectorSgemv(thrust::device_vector<float> A,
                          thrust::device_vector<float> x,
                          thrust::device_vector<float> b,
                          int n, int p,
                          hipblasStatus_t stat, hipblasHandle_t handle)
  {
      float alpha = 1; float beta = 0;
      hipblasSgemv(handle, HIPBLAS_OP_N, n, p, &alpha,
                   thrust::raw_pointer_cast(&A[0]), n,
                   thrust::raw_pointer_cast(&x[0]), 1,
                   &beta, thrust::raw_pointer_cast(&b[0]), 1);
  }

  // S(x, lambda)
  void device_vectorSoftThreshold(thrust::device_vector<float> x,
                                  thrust::device_vector<float> dest,
                                  float lambda)
  {
    thrust::transform(x.begin(), x.end(), dest.begin(), soft_threshold(lambda));
  }

}

int main() {
  int* n = (int*)malloc(sizeof(int)); n[0] = 100;
  int* p = (int*)malloc(sizeof(int)); p[0] = 10;
  int* num_lambda = (int*)malloc(sizeof(int)); num_lambda[0] = 1;
 
  thrust::host_vector<float> X(n[0]*p[0],1);
  thrust::host_vector<float> y(n[0],1);
  thrust::host_vector<float> beta(p[0] * num_lambda[0],1);

  int* type = (int*)malloc(sizeof(int)); type[0] = 0;
  int* maxIt = (int*)malloc(sizeof(int)); maxIt[0] = 10;
  int* reset = (int*)malloc(sizeof(int)); reset[0] = 5;
  float* lambda = (float*)malloc(sizeof(float) * num_lambda[0]); lambda[0] = 1;
  float* thresh = (float*)malloc(sizeof(float)); thresh[0] = 0.0001;
  float* gamma = (float*)malloc(sizeof(float)); gamma[0] = 0.0001;
  float* t = (float*)malloc(sizeof(float)); t[0] = 0.0001;

  activePathSol(thrust::raw_pointer_cast(&X[0]),
                thrust::raw_pointer_cast(&y[0]),
                n, p, lambda, num_lambda,
                type, thrust::raw_pointer_cast(&beta[0]), maxIt, thresh, gamma,
                t, reset);
  
  free(n); free(p); free(num_lambda);
  free(type); free(maxIt); free(reset);
  free(lambda); free(thresh); free(gamma); free(t);
  return 0;
}
